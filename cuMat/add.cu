#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdlib>

#include "add.h"

__global__ void MatAdd(float **A, float **B, float **C){
    int i = threadIdx.x;
    int j = threadIdx.y;
    C[i][j] = A[i][j] + B[i][j];
}

void MatAdd_exec(float **lvalue1, float **lvalue2, float **rvalue){
    float **dev_l1, **dev_l2, **dev_r;

    hipMalloc((void**)&dev_l1, N*N*sizeof(float));
    hipMalloc((void**)&dev_l2, N*N*sizeof(float));
    hipMalloc((void**)&dev_r,  N*N*sizeof(float));

    hipMemcpy(dev_l1, lvalue1, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_l2, lvalue2, N*N*sizeof(float), hipMemcpyHostToDevice);

    int numBlocks = 1;
    dim3 threadsPerBlock(N, N);

    MatAdd<<<numBlocks, threadsPerBlock>>>(dev_l1, dev_l2, dev_r);

    hipMemcpy(rvalue, dev_r, N*N*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_l1);
    hipFree(dev_l2);
    hipFree(dev_r);
}
