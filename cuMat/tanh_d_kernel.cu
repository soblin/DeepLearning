#include "hip/hip_runtime.h"
#include "tanh_d_kernel.h"

#include <cmath>

#define BLOCK_SIZE 32

__device__ __forceinline__ float tanh_d(float a){
    return 1.0 - std::tanh(a)*std::tanh(a);
}

__global__ void tanh_d_kernel(const float *__restrict__ src,
                              float *__restrict__ dst, int m, int n){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x + blockDim.x + threadIdx.x;

    if(row < m && col < n) dst[row * n + col] = tanh_d(src[row * n + col]);
}

void tanh_d_kernel_exec(const float *src, float *dst, int m, int n){
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((n + block.x-1)/block.x, (m + block.y-1)/block.y);

    tanh_d_kernel <<< grid, block >>> (src, dst, m, n);
    hipDeviceSynchronize();
}
