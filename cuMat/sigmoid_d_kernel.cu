#include "hip/hip_runtime.h"
#include "sigmoid_d_kernel.h"

#include <cmath>

#define BLOCK_SIZE 32

__device__ __forceinline__ float sigmoid_d(float a){
    float b = 1.0 / (1.0 + std::exp(-a));
    return b * (1.0 - b);
}

__global__ void sigmoid_d_kernel(const float *__restrict__ src,
                                 float *__restrict__ dst, int m, int n){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < m && col < n) dst[row * n + col] = sigmoid_d(src[row * n + col]);
}

void sigmoid_kernel_exec(const float *src, float *dst, int m, int n){
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((n + block.x-1)/block.x, (m + block.y-1)/block.y);

    sigmoid_d_kernel <<< grid, block >>> (src, dst, m, n);
    hipDeviceSynchronize();
}
