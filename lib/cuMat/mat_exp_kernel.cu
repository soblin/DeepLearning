#include "hip/hip_runtime.h"
#include "DeepLearning/cuMat/mat_exp_kernel.h"
#include <cmath>

static const int block_size = 32;

__device__ __forceinline__ float mat_exp(float a, float alpha){
    return std::exp(a+alpha);
}

__global__ void mat_exp_kernel(const float *__restrict__ src,
                               float *__restrict__ dst, int m, int n, float alpha){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < m && col < n){
        dst[row*n+col] = mat_exp(src[row*n+col], alpha);
    }
}

void mat_exp_kernel_exec(const float *src, float *dst, int m, int n, float alpha){
    dim3 block(block_size, block_size);
    dim3 grid((n+block.x-1)/block.x, (m+block.y-1)/block.y);

    mat_exp_kernel <<< grid, block >>> (src, dst, m, n, alpha);
    hipDeviceSynchronize();
}
