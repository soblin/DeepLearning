#include "hip/hip_runtime.h"
#include "DeepLearning/cuMat/mat_inverse_d_kernel.h"

static const int block_size = 32;

__device__ __forceinline__ float mat_inverse_d(float a){
    return -1.0 / (a+1e-8)*(a+1e-8);
}

__global__ void mat_inverse_d_kernel(const float *__restrict__ src,
                                     float *__restrict__ dst, int m, int n){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < m && col < n){
        dst[row*n+col] = mat_inverse_d(src[row*n+col]);
    }
}

void mat_inverse_d_kernel_exec(const float *src, float *dst, int m, int n){
    dim3 block(block_size, block_size);
    dim3 grid((n+block.x-1)/block.x, (m+block.y-1)/block.y);

    mat_inverse_d_kernel <<< grid, block >>> (src, dst, m, n);
    hipDeviceSynchronize();
}
