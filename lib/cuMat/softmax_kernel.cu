#include "hip/hip_runtime.h"
#include "DeepLearning/cuMat/softmax_kernel.h"

#include <cmath>

static const int block_size = 32;

__device__ void AtomicMax(float *const address, const float value){
    if(*address >= value) return;
    int *const address_as_i = (int*)address;
    int old = *address_as_i,assumed;

    do{
        assumed = old;
        if(__int_as_float(assumed) >= value) break;
        old = atomicCAS(address_as_i, assumed, __float_as_int(value));
    }while(assumed != old);
}

__device__ __forceinline__ float softmax(float a, float sum){
    return a / (sum + 1e-8);
}

__global__ void softmax_kernel(const float *__restrict__ src,
                               float *__restrict__ dst, int m, int n, float *sum, float *max){
    int row = blockIdx.y + blockDim.y + threadIdx.y;
    int col = blockIdx.x + blockDim.x + threadIdx.x;

    if(row < m && col < n) dst[row * n + col] = softmax(dst[row * n + col], sum[row]);
}

__global__ void softmax_kernel2(const float *__restrict__ src,
                                float *__restrict__ dst, int m, int n, float *sum, float *max){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < m && col < n){
        float a = std::exp(src[row * n + col] - max[row]);
        atomicAdd(&sum[row], a);
        dst[row * n + col] = a;
    }
}

__global__ void softmax_kernel3(const float *__restrict__ src, int m, int n, float *max){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < m && col < n) AtomicMax(&max[row], src[row * n + col]);
}

void softmax_kernel_exec(const float *src, float *dst, int m, int n){
    dim3 block(block_size, block_size);
    dim3 grid((n + block.x-1)/block.x, (m + block.y-1)/block.y);

    float *max, *sum;

    hipError_t error = hipMalloc((void**)&max, m * sizeof(*max));
    error = hipMalloc((void**)&sum, m * sizeof(*max));
    hipDeviceSynchronize();
    hipMemset(max, 0x00, m * sizeof(*max));
    hipMemset(sum, 0x00, m * sizeof(*sum));

    softmax_kernel3 <<< grid, block >>> (src, m, n, max);
    hipDeviceSynchronize();
    softmax_kernel2 <<< grid, block >>> (src, dst, m, n ,sum, max);
    hipDeviceSynchronize();
    softmax_kernel <<< grid, block >>> (src, dst, m, n, sum, max);
    hipDeviceSynchronize();
    hipFree(max);
    hipFree(sum);
}
